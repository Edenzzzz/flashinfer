/*
 * Copyright (c) 2023 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/attention/variants.cuh>
#include <flashinfer/pos_enc.cuh>
#include <optional>

#include "pod_config.inc"
#include "pytorch_conversion_utils.h"
#include "pytorch_extension_utils.h"

namespace flashinfer {
template <uint32_t HEAD_DIM_QK, uint32_t HEAD_DIM_VO, PosEncodingMode POS_ENCODING_MODE,
          bool USE_FP16_QK_REDUCTION, MaskMode MASK_MODE_P, uint32_t CTA_TILE_Q_P,
          uint32_t CTA_TILE_Q_D, MaskMode MASK_MODE_D, typename PrefillAttentionVariant,
          typename DecodeAttentionVariant, typename PrefillParams, typename DecodeParams>
hipError_t PODWithKVCacheTensorDispatched(PrefillParams prefill_params, DecodeParams decode_params,
                                           typename DecodeParams::DTypeO* tmp_v, float* tmp_s,
                                           bool enable_pdl, hipStream_t stream);

}  // namespace flashinfer

using namespace flashinfer;

at::Tensor PODWithKVCachePlan(at::Tensor float_workspace_buffer, at::Tensor int_workspace_buffer,
                              at::Tensor page_locked_int_workspace_buffer, at::Tensor qo_indptr_p,
                              at::Tensor kv_indptr_p, at::Tensor kv_len_arr_p,
                              uint32_t total_num_rows_p, uint32_t batch_size_p,
                              at::Tensor qo_indptr_d, at::Tensor kv_indptr_d,
                              uint32_t total_num_rows_d, uint32_t batch_size_d,
                              uint32_t num_qo_heads_p, uint32_t num_kv_heads, uint32_t head_dim_qk,
                              uint32_t head_dim_vo, uint32_t page_size, bool enable_cuda_graph) {
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();

  PODPlanInfo plan_info;

  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer.device());
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();
  hipError_t status =
      PODPlan<IdType>(float_workspace_buffer.data_ptr(), float_workspace_size_in_bytes,
                      int_workspace_buffer.data_ptr(), page_locked_int_workspace_buffer.data_ptr(),
                      int_workspace_size_in_bytes, plan_info, qo_indptr_p.data_ptr<IdType>(),
                      kv_indptr_p.data_ptr<IdType>(), total_num_rows_p, batch_size_p,
                      qo_indptr_d.data_ptr<IdType>(), kv_indptr_d.data_ptr<IdType>(),
                      total_num_rows_d, batch_size_d, num_qo_heads_p, num_kv_heads, head_dim_qk,
                      head_dim_vo, page_size, enable_cuda_graph, /*sizeof_dtype_o=*/2, stream);

  TORCH_CHECK(status == hipSuccess,
              "Failed to plan prefill with error: ", hipGetErrorString(status));

  return vec_to_tensor(plan_info.ToVector());
}

void PODWithKVCacheTensorRun(
    // Shared params
    at::Tensor float_workspace_buffer_d, at::Tensor int_workspace_buffer_d,
    at::Tensor plan_info_vec, at::Tensor paged_k_cache, at::Tensor paged_v_cache,
    at::Tensor qo_indptr, at::Tensor paged_kv_indptr, at::Tensor paged_kv_indices,
    at::Tensor paged_kv_last_page_len, at::Tensor o, std::optional<at::Tensor> maybe_lse,
    int64_t layout,
    // Prefill params
    at::Tensor q_p, int64_t mask_mode_code_p, int64_t window_left_p,
    std::optional<at::Tensor> maybe_custom_mask_p, std::optional<at::Tensor> maybe_alibi_slopes_p,
    double logits_soft_cap_p, double sm_scale_p, double rope_rcp_scale_p, double rope_rcp_theta_p,
    // Decode params
    at::Tensor q_d, int64_t mask_mode_code_d, int64_t window_left_d,
    std::optional<at::Tensor> maybe_custom_mask_d, std::optional<at::Tensor> maybe_mask_indptr_d,
    std::optional<at::Tensor> maybe_alibi_slopes_d, double logits_soft_cap_d, double sm_scale_d,
    double rope_rcp_scale_d, double rope_rcp_theta_d, bool enable_pdl) {
  PODPlanInfo plan_info;
  plan_info.FromVector(tensor_to_vec(plan_info_vec));
  auto device = q_d.device();
  uint32_t batch_size = paged_kv_indptr.size(0) - 1;
  void* float_buffer_ptr = static_cast<void*>(float_workspace_buffer_d.data_ptr());
  void* int_buffer_ptr = static_cast<void*>(int_workspace_buffer_d.data_ptr());
  // get kv_cache_strides
  const int64_t* kv_cache_strides = nullptr;
  auto k_strides = paged_k_cache.strides();
  auto v_strides = paged_v_cache.strides();
  TORCH_CHECK(k_strides == v_strides, "k/v strides must be identical");
  kv_cache_strides = k_strides.data();

  // Prefill setup
  uint32_t head_dim_qk = q_p.size(2);
  uint32_t qo_len, num_qo_heads_p;
  QKVLayout kv_layout = static_cast<QKVLayout>(layout);
  qo_len = q_p.size(0) + q_d.size(0);
  num_qo_heads_p = q_p.size(1);
  uint32_t q_stride_n_p = q_p.stride(0), q_stride_h_p = q_p.stride(1);
  if (maybe_lse) {
    const auto& lse = *maybe_lse;
    TORCH_CHECK(lse.size(0) == qo_len, lse.size(0), qo_len);
    TORCH_CHECK(lse.size(1) == num_qo_heads_p, lse.size(1), q_p.size(1));
  }

  const MaskMode mask_mode_p = static_cast<MaskMode>(mask_mode_code_p);

  auto q_scalar_type = q_p.scalar_type();

  // Decode setup (Tensor decode = batched prefill)
  uint32_t num_qo_heads = q_d.size(1);
  TORCH_CHECK(num_qo_heads_p == num_qo_heads,
              "POD currently requires same # Query heads for prefill and decode");

  uint32_t num_kv_heads_d, num_kv_heads, page_size;
  if (kv_layout == QKVLayout::kHND) {
    num_kv_heads = paged_k_cache.size(1);
    num_kv_heads_d = paged_k_cache.size(1);
    page_size = paged_k_cache.size(2);
  } else {
    num_kv_heads = paged_k_cache.size(2);
    num_kv_heads_d = paged_k_cache.size(2);
    page_size = paged_k_cache.size(1);
  }
  TORCH_CHECK(num_kv_heads == num_kv_heads_d,
              "POD currently requires same # KV heads for prefill and decode; Prefill: ",
              num_kv_heads, ", Decode: ", num_kv_heads_d);

  const MaskMode mask_mode_d = static_cast<MaskMode>(mask_mode_code_d);

  // get q_stride_n and q_stride_h
  const auto q_stride_n_d = q_d.stride(0);
  const auto q_stride_h_d = q_d.stride(1);

  const c10::cuda::OptionalCUDAGuard device_guard(float_workspace_buffer_d.device());
  const hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  DISPATCH_context(
      MASK_MODE_P, MASK_MODE_D, DTypeQ, DTypeKV, HEAD_DIM_QK, USE_SLIDING_WINDOW_P,
      USE_SLIDING_WINDOW_D, USE_LOGITS_SOFT_CAP, [&] {
        paged_kv_t<DTypeKV, IdType> paged_kv(
            num_kv_heads, page_size, HEAD_DIM_VO, batch_size, kv_layout,
            static_cast<DTypeKV*>(paged_k_cache.data_ptr()),
            static_cast<DTypeKV*>(paged_v_cache.data_ptr()), kv_cache_strides,
            static_cast<IdType*>(paged_kv_indices.data_ptr()),
            static_cast<IdType*>(paged_kv_indptr.data_ptr()),
            static_cast<IdType*>(paged_kv_last_page_len.data_ptr()));
        PrefillParams prefill_params;
        {
          // Make params a reference to prefill_params to set values
          PrefillParams& params = prefill_params;
          params.q = static_cast<DTypeQ*>(q_p.data_ptr());
          params.paged_kv = paged_kv;
          params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());
          params.o = static_cast<DTypeO*>(o.data_ptr());
          params.lse = maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
          params.group_size = uint_fastdiv(num_qo_heads / paged_kv.num_heads);
          params.q_stride_n = q_stride_n_p;
          params.q_stride_h = q_stride_h_p;
          params.window_left = window_left_p;
          params.paged_kv.num_heads = num_kv_heads;
          params.num_qo_heads = num_qo_heads;

          params.request_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.request_indices_offset);
          params.qo_tile_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.qo_tile_indices_offset);
          params.kv_tile_indices =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_tile_indices_offset);
          params.o_indptr = GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.o_indptr_offset);
          if (plan_info.split_kv) {
            params.merge_indptr =
                GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.merge_indptr_offset);
            if (plan_info.enable_cuda_graph) {
              params.block_valid_mask =
                  GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
            }
          }
          params.kv_chunk_size_ptr =
              GetPtrFromBaseOffset<IdType>(int_buffer_ptr, plan_info.kv_chunk_size_ptr_offset_p);
          params.padded_batch_size = plan_info.padded_batch_size_p;
          params.maybe_custom_mask = maybe_custom_mask_p
                                         ? static_cast<uint8_t*>(maybe_custom_mask_p->data_ptr())
                                         : nullptr;
          params.maybe_alibi_slopes = maybe_alibi_slopes_p
                                          ? static_cast<float*>(maybe_alibi_slopes_p->data_ptr())
                                          : nullptr;
          params.logits_soft_cap = logits_soft_cap_p;
          params.sm_scale = sm_scale_p;
          params.rope_rcp_scale = rope_rcp_scale_p;
          params.rope_rcp_theta = rope_rcp_theta_p;
          params.max_total_num_rows = plan_info.total_num_rows;
          if (plan_info.enable_cuda_graph) {
            params.total_num_rows =
                GetPtrFromBaseOffset<uint32_t>(int_buffer_ptr, plan_info.total_num_rows_offset);
          }
          params.partition_kv = plan_info.split_kv;
          if (plan_info.split_kv) {
            if (plan_info.enable_cuda_graph) {
              params.block_valid_mask =
                  GetPtrFromBaseOffset<bool>(int_buffer_ptr, plan_info.block_valid_mask_offset);
            }
          }
        }

        DecodeParams decode_params;
        DTypeO* tmp_v = nullptr;
        float* tmp_s = nullptr;
        {
          DecodeParams& params = decode_params;
          params.q = static_cast<DTypeQ*>(q_d.data_ptr());
          params.paged_kv = paged_kv;
          params.q_indptr = static_cast<IdType*>(qo_indptr.data_ptr());
          params.o = static_cast<DTypeO*>(o.data_ptr());
          params.lse = maybe_lse ? static_cast<float*>(maybe_lse->data_ptr()) : nullptr;
          params.group_size = uint_fastdiv(num_qo_heads / paged_kv.num_heads);
          params.q_stride_n = q_stride_n_d;
          params.q_stride_h = q_stride_h_d;
          params.window_left = window_left_d;
          params.paged_kv.num_heads = num_kv_heads;
          params.num_qo_heads = num_qo_heads;

          params.request_indices = prefill_params.request_indices;
          params.qo_tile_indices = prefill_params.qo_tile_indices;
          params.kv_tile_indices = prefill_params.kv_tile_indices;
          params.o_indptr = prefill_params.o_indptr;
          params.kv_chunk_size_ptr = prefill_params.kv_chunk_size_ptr;

          params.partition_kv = plan_info.split_kv;
          if (plan_info.split_kv) {
            params.merge_indptr = prefill_params.merge_indptr;
            // These should be assigned from plan info, not from prefill_params
            tmp_v = GetPtrFromBaseOffset<DTypeO>(float_buffer_ptr, plan_info.v_offset);
            tmp_s = GetPtrFromBaseOffset<float>(float_buffer_ptr, plan_info.s_offset);
            if (plan_info.enable_cuda_graph) {
              params.block_valid_mask = prefill_params.block_valid_mask;
            }
          }
          params.padded_batch_size = plan_info.padded_batch_size_d;
          params.max_total_num_rows = plan_info.total_num_rows;

          params.maybe_mask_indptr = maybe_mask_indptr_d
                                         ? static_cast<int32_t*>(maybe_mask_indptr_d->data_ptr())
                                         : nullptr;
          params.maybe_alibi_slopes = maybe_alibi_slopes_d
                                          ? static_cast<float*>(maybe_alibi_slopes_d->data_ptr())
                                          : nullptr;
          params.logits_soft_cap = logits_soft_cap_d;
          params.sm_scale = sm_scale_d;
          params.rope_rcp_scale = rope_rcp_scale_d;
          params.rope_rcp_theta = rope_rcp_theta_d;

          if (plan_info.enable_cuda_graph) {
            params.total_num_rows = prefill_params.total_num_rows;
          }
        }

        constexpr bool use_custom_mask_p = MASK_MODE_P == MaskMode::kCustom;
        using PrefillAttentionVariant =
            DefaultAttention</*use_custom_mask=*/use_custom_mask_p, USE_SLIDING_WINDOW_P,
                             USE_LOGITS_SOFT_CAP, /*use_alibi_bias=*/false>;
        constexpr bool use_custom_mask_d = MASK_MODE_D == MaskMode::kCustom;
        using DecodeAttentionVariant =
            DefaultAttention</*use_custom_mask=*/use_custom_mask_d, USE_SLIDING_WINDOW_D,
                             USE_LOGITS_SOFT_CAP, /*use_alibi_bias=*/false>;
        // DISPATCH_CTA_TILE_Q(plan_info.cta_tile_q, CTA_TILE_Q, {
        constexpr size_t CTA_TILE_Q_P = plan_info.cta_tile_q_p;
        constexpr size_t CTA_TILE_Q_D = plan_info.cta_tile_q_d;
        hipError_t status = flashinfer::PODWithKVCacheTensorDispatched<
            HEAD_DIM_QK, HEAD_DIM_VO, POS_ENCODING_MODE, USE_FP16_QK_REDUCTION, MASK_MODE_P,
            CTA_TILE_Q_P, CTA_TILE_Q_D, MASK_MODE_D, PrefillAttentionVariant,
            DecodeAttentionVariant>(prefill_params, decode_params, tmp_v, tmp_s, enable_pdl,
                                    stream);
        TORCH_CHECK(status == hipSuccess, "PODWithKVCache kernel launch failed, error: " +
                                               std::string(hipGetErrorString(status)));
        //});
      });
}
